#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h> 
#include "BlochKernelGMGPU.h" // Your kernel file

#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

float *d_b_Mx, *d_b_My, *d_b_Mz;
float  *d_Buffer;
float *d_K, *d_TypeFlag;
float totalSpins, SBufferLen, SignalLen, SeqLen, RxCoilNum, TxCoilNum, TypeNum, SpinNum, SpinMxZ, SpinMxY, SpinMxX;


int main() {

    float *d_Mz = NULL;
    hipMalloc( (void**) &d_Mz, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float)) ;
	hipMemcpy( d_Mz, Mz, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice ) ;
    
    float *d_My = NULL;
    hipMalloc( (void**) &d_My, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float)) ;
	hipMemcpy( d_My, My, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice ) ;
    
    float *d_Mx = NULL;
    hipMalloc( (void**) &d_Mx, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float)) ;
	hipMemcpy( d_Mx, Mx, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice ) ;
    
    float *d_dWRnd = NULL;
    hipMalloc( (void**) &d_dWRnd, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float)) ;
	hipMemcpy( d_dWRnd, dWRnd, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice ) ;
    
    float *d_Rho = NULL;
    hipMalloc( (void**) &d_Rho, SpinMxNum * SpinMxSliceNum * (*TypeNum) * sizeof(float)) ;
	hipMemcpy( d_Rho, Rho, SpinMxNum * SpinMxSliceNum * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice ) ;
    
    float *d_T1 = NULL;
    hipMalloc( (void**) &d_T1, SpinMxNum * SpinMxSliceNum * (*TypeNum) * sizeof(float)) ;
	hipMemcpy( d_T1, T1, SpinMxNum * SpinMxSliceNum * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice ) ;
    
    float *d_T2 = NULL;
    hipMalloc( (void**) &d_T2, SpinMxNum * SpinMxSliceNum * (*TypeNum) * sizeof(float)) ;
	hipMemcpy( d_T2, T2, SpinMxNum * SpinMxSliceNum * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice ) ;
    
    float *d_Gzgrid = NULL;
    hipMalloc( (void**) &d_Gzgrid, SpinMxNum * SpinMxSliceNum * sizeof(float)) ;
	hipMemcpy( d_Gzgrid, Gzgrid, SpinMxNum * SpinMxSliceNum * sizeof(float), hipMemcpyHostToDevice ) ;
    
    float *d_Gygrid = NULL;
    hipMalloc( (void**) &d_Gygrid, SpinMxNum * SpinMxSliceNum * sizeof(float)) ;
	hipMemcpy( d_Gygrid, Gygrid, SpinMxNum * SpinMxSliceNum * sizeof(float), hipMemcpyHostToDevice ) ;
    
    float *d_Gxgrid = NULL;
    hipMalloc( (void**) &d_Gxgrid, SpinMxNum * SpinMxSliceNum * sizeof(float)) ;
	hipMemcpy( d_Gxgrid, Gxgrid, SpinMxNum * SpinMxSliceNum * sizeof(float), hipMemcpyHostToDevice ) ;
    
    float *d_dB0 = NULL;
    hipMalloc( (void**) &d_dB0, SpinMxNum * SpinMxSliceNum * sizeof(float)) ;
	hipMemcpy( d_dB0, dB0, SpinMxNum * SpinMxSliceNum * sizeof(float), hipMemcpyHostToDevice ) ;

    float *d_TxCoilmg = NULL;
    hipMalloc( (void**) &d_TxCoilmg, SpinMxNum * SpinMxSliceNum * (*TxCoilNum) * sizeof(float)) ;
	hipMemcpy( d_TxCoilmg, TxCoilmg, SpinMxNum * SpinMxSliceNum * (*TxCoilNum) * sizeof(float), hipMemcpyHostToDevice ) ;

    float *d_TxCoilpe = NULL;
    hipMalloc( (void**) &d_TxCoilpe, SpinMxNum * SpinMxSliceNum * (*TxCoilNum) * sizeof(float)) ;
	hipMemcpy( d_TxCoilpe, TxCoilpe, SpinMxNum * SpinMxSliceNum * (*TxCoilNum) * sizeof(float), hipMemcpyHostToDevice ) ;
	
	float *d_RxCoilx = NULL;
    hipMalloc( (void**) &d_RxCoilx, SpinMxNum * SpinMxSliceNum * (*RxCoilNum) * sizeof(float)) ;
	hipMemcpy( d_RxCoilx, RxCoilx, SpinMxNum * SpinMxSliceNum * (*RxCoilNum) * sizeof(float), hipMemcpyHostToDevice ) ;

	float *d_RxCoily = NULL;
    hipMalloc( (void**) &d_RxCoily, SpinMxNum * SpinMxSliceNum * (*RxCoilNum) * sizeof(float)) ;
	hipMemcpy( d_RxCoily, RxCoily, SpinMxNum * SpinMxSliceNum * (*RxCoilNum) * sizeof(float), hipMemcpyHostToDevice ) ;
	
    double *d_CS = NULL;
    hipMalloc( (void**) &d_CS, *TypeNum * sizeof(double)) ;
	hipMemcpy( d_CS, CS, *TypeNum * sizeof(double), hipMemcpyHostToDevice ) ;
	
    /* allocate device memory for GPU execution sequence*/
    float *d_Sig = NULL;
    hipMalloc( (void**) &d_Sig, (5+3*(*TxCoilNum)) * MaxutsStep * sizeof(float)) ;

    /**/
    float *d_Sx = NULL;
    hipMalloc( (void**) &d_Sx, SpinMxNum * PreSignalLen * (*TypeNum) * (*RxCoilNum) * sizeof(float)) ;
    float *d_Sy = NULL;
    hipMalloc( (void**) &d_Sy, SpinMxNum * PreSignalLen * (*TypeNum) * (*RxCoilNum) * sizeof(float)) ;

    SpinMxX = 0;
    SpinMxY = 0;
    SpinMxZ = 0;
    SpinNum = 12;
    TypeNum = 10;
    TxCoilNum = 125;
    RxCoilNum = 122;
    SeqLen = 25;
    SignalLen = 24;
    SBufferLen = 100;

    totalSpins = SpinMxX * SpinMxY * SpinMxZ * SpinNum * TypeNum;

    CHECK_CUDA(hipMalloc(&d_Mx, sizeof(float) * totalSpins));
    CHECK_CUDA(hipMalloc(&d_My, sizeof(float) * totalSpins));
    CHECK_CUDA(hipMalloc(&d_Mz, sizeof(float) * totalSpins));

    CHECK_CUDA(hipMalloc(&d_Rho, sizeof(float) * totalSpins));
    CHECK_CUDA(hipMalloc(&d_T1, sizeof(float) * totalSpins));
    CHECK_CUDA(hipMalloc(&d_T2, sizeof(float) * totalSpins));
    CHECK_CUDA(hipMalloc(&d_K, sizeof(float) * totalSpins));
    CHECK_CUDA(hipMalloc(&d_dB0, sizeof(float) * totalSpins));
    CHECK_CUDA(hipMalloc(&d_dWRnd, sizeof(float) * totalSpins));

    // Dummy values
    float Gyro = 42.58e6f;
    float RxCoilDefault = 1.0f, TxCoilDefault = 1.0f;
    float rfRef = 0;



    CHECK_CUDA(hipMalloc(&d_CS, sizeof(double) * TypeNum));
    CHECK_CUDA(hipMalloc(&d_TypeFlag, sizeof(double) * TypeNum));
    CHECK_CUDA(hipMalloc(&d_Buffer, sizeof(float) * 6 * TypeNum * SpinMxX * SpinMxY));
    CHECK_CUDA(hipMalloc(&d_Sig, sizeof(float) * SeqLen * (5 + 3 * TxCoilNum)));
    CHECK_CUDA(hipMalloc(&d_Sx, sizeof(float) * SignalLen));
    CHECK_CUDA(hipMalloc(&d_Sy, sizeof(float) * SignalLen));
    CHECK_CUDA(hipMalloc(&d_Gzgrid, sizeof(float) * totalSpins));
    CHECK_CUDA(hipMalloc(&d_Gygrid, sizeof(float) * totalSpins));
    CHECK_CUDA(hipMalloc(&d_Gxgrid, sizeof(float) * totalSpins));
    CHECK_CUDA(hipMalloc(&d_TxCoilmg, sizeof(float) * totalSpins));
    CHECK_CUDA(hipMalloc(&d_TxCoilpe, sizeof(float) * totalSpins));
    CHECK_CUDA(hipMalloc(&d_RxCoilx, sizeof(float) * totalSpins));
    CHECK_CUDA(hipMalloc(&d_RxCoily, sizeof(float) * totalSpins));
    CHECK_CUDA(hipMalloc(&d_b_Mx, sizeof(float) * totalSpins));
    CHECK_CUDA(hipMalloc(&d_b_My, sizeof(float) * totalSpins));
    CHECK_CUDA(hipMalloc(&d_b_Mz, sizeof(float) * totalSpins));

    // Launch CUDA kernel
    dim3 blockDim(1, 1, 1);
    dim3 gridDim(1, 1, 1);
    size_t sharedMemSize = SeqLen * (5 + 3 * TxCoilNum) * sizeof(float);

    BlochKernelNormalGPU<<<gridDim, blockDim, sharedMemSize>>>(
         Gyro,  d_CS,  d_Rho,  d_T1, d_T2,  d_Mz,  d_My,  d_Mx,
		d_dB0,  d_dWRnd,  d_Gzgrid,  d_Gygrid,  d_Gxgrid,  d_TxCoilmg,  d_TxCoilpe,  d_RxCoilx,  d_RxCoily, 
		d_Sig,  RxCoilDefault,  TxCoilDefault,
		d_Sx,  d_Sy,  rfRef,  SignalLen,  SBufferLen,
		SpinMxX,  SpinMxY,  SpinMxZ,  SpinNum,  TypeNum,  TxCoilNum,  RxCoilNum,  SeqLen
    );

    CHECK_CUDA(hipDeviceSynchronize());

    std::cout << "Kernel executed successfully!" << std::endl;

    // Free memory
    hipFree(d_Mx); hipFree(d_My); hipFree(d_Mz);
    hipFree(d_Rho); hipFree(d_T1); hipFree(d_T2); hipFree(d_K);
    hipFree(d_dB0); hipFree(d_dWRnd);
    hipFree(d_CS); hipFree(d_TypeFlag); hipFree(d_Buffer); hipFree(d_Sig);
    hipFree(d_Sx); hipFree(d_Sy);
    hipFree(d_Gzgrid); hipFree(d_Gygrid); hipFree(d_Gxgrid);
    hipFree(d_TxCoilmg); hipFree(d_TxCoilpe);
    hipFree(d_RxCoilx); hipFree(d_RxCoily);
    hipFree(d_b_Mx); hipFree(d_b_My); hipFree(d_b_Mz);

    return 0;
}
