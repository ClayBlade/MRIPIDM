#include "hip/hip_runtime.h"


/************************************************************************
 MEX code for spin discrete evolution using IPP or Framewave and 
 parallel GPU computation (CUDA) written by Fang Liu (leoliuf@gmail.com).
************************************************************************/

/* system header */
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <vector>

/* nVIDIA CUDA header */
#include <hip/hip_runtime.h> 
/* OpenMP header*/
#include <omp.h>
/* Intel IPP header */
#ifdef IPP
#include <ipp.h>
#endif
/* AMD Framewave header */
#ifdef FW
#include <fwSignal.h>
#include <fwBase.h>
#define Ipp32f                  Fw32f
#define ippAlgHintFast          fwAlgHintFast
#define ippsMalloc_32f          fwsMalloc_32f
#define ippsFree                fwsFree
#define ippsZero_32f            fwsZero_32f
#define ippsZero_64f            fwsZero_64f
#define ippsSum_32f             fwsSum_32f
#define ippsCopy_32f            fwsCopy_32f
#define ippsAddC_32f            fwsAddC_32f
#define ippsAddC_32f_I          fwsAddC_32f_I
#define ippsAdd_32f             fwsAdd_32f 
#define ippsAdd_32f_I           fwsAdd_32f_I
#define ippsMulC_32f            fwsMulC_32f
#define ippsMulC_32f_I          fwsMulC_32f_I
#define ippsMul_32f             fwsMul_32f
#define ippsMul_32f_I           fwsMul_32f_I
#define ippsDiv_32f             fwsDiv_32f
#define ippsDivC_32f            fwsDivC_32f
#define ippsInv_32f_A24         fwsInv_32f_A24
#define ippsThreshold_LT_32f_I  fwsThreshold_LT_32f_I
#define ippsExp_32f_I           fwsExp_32f_I
#define ippsArctan_32f          fwsArctan_32f
#define ippsSqr_32f             fwsSqr_32f
#define ippsSqr_32f_I           fwsSqr_32f_I
#define ippsSqrt_32f_I          fwsSqrt_32f_I
#define ippsSin_32f_A24         fwsSin_32f_A24
#define ippsCos_32f_A24         fwsCos_32f_A24
#define ippsPolarToCart_32f     fwsPolarToCart_32f
#define ippsCartToPolar_32f     fwsCartToPolar_32f
#endif

/* for fixing error : identifier "IUnknown" is undefined" */
#ifdef _WIN32
#define WIN32_LEAN_AND_MEAN
#endif

#if defined(_WIN32) || defined(_WIN64)
#include <windows.h>
#endif

#define PI      3.14159265359 /* pi constant */

#include "BlochKernelGMGPU.h"
#include "json.hpp"

using json = nlohmann::json; 

int main(){
    std::ifstream inputFile("/root/output/labeledSpaceJSON/1.pkl.json");

    json data_obj;
    inputFile >> data_obj; 

/* pointers for VObj */
    double *Gyro;
    int SpinMxNum, SpinMxColNum, SpinMxRowNum, SpinMxSliceNum, SpinMxDimNum;
    const mwSize *SpinMxDims;
	float *Mz, *My, *Mx, *Rho, *T1, *T2;

/* pointers for VMag */
    float *dB0, *dWRnd, *Gzgrid, *Gygrid, *Gxgrid;
    
/* pointers for VCoi */
    float *RxCoilx, *RxCoily, *TxCoilmg, *TxCoilpe;
	double *RxCoilDefault, *TxCoilDefault;
    
/* pointers for VCtl */
    double *CS;
    int *TRNum, *MaxThreadNum, ThreadNum;
	int *ActiveThreadNum;
	int *GPUIndex;
    
/* pointers for VSeq */
    double *utsLine, *tsLine, *rfAmpLine, *rfPhaseLine, *rfFreqLine, *rfCoilLine, *GzAmpLine, *GyAmpLine, *GxAmpLine, *ADCLine, *ExtLine, *flagsLine;

/* pointers for VVar */
    double *t, *dt, *rfAmp, *rfPhase, *rfFreq, *rfCoil, *rfRef, *GzAmp, *GyAmp, *GxAmp, *ADC, *Ext, *KzTmp, *KyTmp, *KxTmp, *gpuFetch;
    int *utsi, *rfi, *Gzi, *Gyi, *Gxi, *ADCi, *Exti, *TRCount;
    
/* pointers for VSig */
    double *Sx, *Sy, *Kz, *Ky, *Kx;
	double *p_Sx, *p_Sy;
	
/* loop control */
    int i=0, j=0, s=0, Signali=0, Signalptr=0, PreSignalLen=0, SignalLen=0, SBufferLen=0, Typei, RxCoili, TxCoili;
    int MaxStep, MaxutsStep, MaxrfStep, MaxGzStep, MaxGyStep, MaxGxStep, *SpinNum, *TypeNum, *TxCoilNum, *RxCoilNum, *SignalNum;
    double flag[6];
    
/* IPP or FW buffer */
    Ipp32f buffer, *Sxbuffer, *Sybuffer;
	
/* function status */
    int ExtCall;
    
/* GPU execution sequence */
	std::vector<float> g_Sig;	

    
/* assign pointers */
    /*VObj*/
   Gyro             = (float*) 2.67e08;
    Mz              = (float*) data_obj["Mz"];
    My              = (float*) data_obj["My"];
    Mx              = (float*) data_obj["Mx"];
    Rho             = (float*) data_obj["Rho"];
    T1              = (float*) data_obj["T1"];
    T2              = (float*) data_obj["T2"];
    SpinNum         = (int*)   data_obj["xSize"] * data_obj["ySize"];
    TypeNum         = (int*) 1;

/*DoScanAtGPU.cu(140): error: expression must have arithmetic or unscoped enum type
      dWRnd = (float*) j["xSize"] * (int*) j["ySize"] * (int*) j["zSize"] * SpinNum * TypeNum
*/

    /*VMag*/
    dB0             = (float*)( data_obj["xSize"] * data_obj["ySize"] * data_obj["zSize"]);
    dWRnd           = (float*) (data_obj["xSize"] * data_obj["ySize"] * data_obj["zSize"] * SpinNum * TypeNum);
    Gzgrid          = (float*) (data_obj["xSize"] * data_obj["ySize"] * data_obj["zSize"]);
    Gygrid          = (float*) (data_obj["xSize"] * data_obj["ySize"] * data_obj["zSize"]);
    Gxgrid          = (float*) (data_obj["xSize"] * data_obj["ySize"] * data_obj["zSize"]);
    
    /*VCoi*/
    TxCoilmg        = (float*) 0;
    TxCoilpe        = (float*) 0;
    RxCoilx         = (float*) 0;
    RxCoily         = (float*) 0;
    TxCoilNum       = (int*)   0;
    RxCoilNum       = (int*)   0;
	TxCoilDefault   = (double*) 1;
    RxCoilDefault   = (double*) 1;
     
    /*VCtl*/
    CS              = (double*) 0;
    TRNum  			= (int*)    0;
    MaxThreadNum    = (int*)    0;
	ActiveThreadNum = (int*)    0;
	GPUIndex		= (int*)    0;
    
    /*VSeq*/
    utsLine         = (double*) 0;
    tsLine          = (double*) 0;
    rfAmpLine       = (double*) 0;
    rfPhaseLine     = (double*) 0;
    rfFreqLine      = (double*) 0;
    rfCoilLine      = (double*) 0;
    GzAmpLine       = (double*) 0;
    GyAmpLine       = (double*) 0;
    GxAmpLine       = (double*) 0;
    ADCLine         = (double*) 0;
    ExtLine         = (double*) 0;
    flagsLine       = (double*) 0;
    
    
    MaxStep         = 0;
    MaxutsStep      = 0;
    MaxrfStep       = 0;
    MaxGzStep       = 0;
    MaxGyStep       = 0;
    MaxGxStep       = 0;
	
    /*VVar*/
	t               = (double*) 0;
    dt              = (double*) 0;
    rfAmp           = (double*) 0;
    rfPhase         = (double*) 0;
    rfFreq          = (double*) 0;
    rfCoil          = (double*) 0;
    rfRef           = (double*) 0;
    GzAmp           = (double*) 0;
    GyAmp           = (double*) 0;
    GxAmp           = (double*) 0;
    ADC             = (double*) 0;
    Ext             = (double*) 0;
    KzTmp           = (double*) 0;
    KyTmp           = (double*) 0;
    KxTmp           = (double*) 0;
	gpuFetch     	= (double*) 0;
    utsi            = (int*)    0;
    rfi             = (int*)    0;
    Gzi             = (int*)    0;
    Gyi             = (int*)    0;
    Gxi             = (int*)    0;
    ADCi            = (int*)	0;
    Exti            = (int*)    0;
    TRCount         = (int*)    0;
	
    /*VSig*/ 
	Sy              = (double*) 0;
    Sx              = (double*) 0;
    Kz              = (double*) 0;
    Ky              = (double*) 0;
    Kx              = (double*) 0;
    SignalNum       = (int*)    0;
    
/* get size of spin matrix */
    SpinMxDimNum    		= 3;
    /*SpinMxDims      		= (mwSize*) mxCalloc(SpinMxDimNum, sizeof(mwSize));*/

    size_t *SpinMxDims = (size_t*) malloc(SpinNum * sizeof(size_t));
    SpinMxDims[0] = (mwSize*) data_obj["xSize"];
    SpinMxDims[1] = (mwSize*) data_obj["ySize"]; 
    SpinMxDims[2] = (mwSize*) data_obj["zSize"];
    /*Might be y by x by z*/
	
    SpinMxRowNum    		= SpinMxDims[0];
    SpinMxColNum    		= SpinMxDims[1];
    SpinMxNum       		= SpinMxDims[0] * SpinMxDims[1];
    if (SpinMxDimNum == 2){
        SpinMxSliceNum = 1;
    }else{
        SpinMxSliceNum = SpinMxDims[2];
    }
	
/* choose selected GPU */
	if( hipSuccess != hipSetDevice(*GPUIndex)){
        return;
    }
	
/* set GPU grid & block configuration*/
    hipDeviceProp_t deviceProp;
    memset( &deviceProp, 0, sizeof(deviceProp));
    if( hipSuccess != hipGetDeviceProperties(&deviceProp, *GPUIndex)){
        return;
    }

	dim3 dimGridImg(SpinMxColNum,1,1);
    dim3 dimBlockImg(1,SpinMxRowNum,1);

	for (i=SpinMxColNum - 1; i >= deviceProp.multiProcessorCount; i--){
		if ( SpinMxNum % i == 0 ){
			if (SpinMxNum/i > deviceProp.maxThreadsPerBlock) break;
			if ((SpinMxNum/i)*63 > deviceProp.regsPerBlock) break; // 63 registers per thread for current kernel
			dimGridImg.x = i;
		    dimBlockImg.y = SpinMxNum/i;
		}
	}
	i=0;
	
/* allocate device memory for matrices */
    float *d_Mz = NULL;
    hipMalloc( (void**) &d_Mz, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float)) ;
	hipMemcpy( d_Mz, Mz, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice ) ;
    
    float *d_My = NULL;
    hipMalloc( (void**) &d_My, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float)) ;
	hipMemcpy( d_My, My, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice ) ;
    
    float *d_Mx = NULL;
    hipMalloc( (void**) &d_Mx, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float)) ;
	hipMemcpy( d_Mx, Mx, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice ) ;
    
    float *d_dWRnd = NULL;
    hipMalloc( (void**) &d_dWRnd, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float)) ;
	hipMemcpy( d_dWRnd, dWRnd, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice ) ;
    
    float *d_Rho = NULL;
    hipMalloc( (void**) &d_Rho, SpinMxNum * SpinMxSliceNum * (*TypeNum) * sizeof(float)) ;
	hipMemcpy( d_Rho, Rho, SpinMxNum * SpinMxSliceNum * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice ) ;
    
    float *d_T1 = NULL;
    hipMalloc( (void**) &d_T1, SpinMxNum * SpinMxSliceNum * (*TypeNum) * sizeof(float)) ;
	hipMemcpy( d_T1, T1, SpinMxNum * SpinMxSliceNum * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice ) ;
    
    float *d_T2 = NULL;
    hipMalloc( (void**) &d_T2, SpinMxNum * SpinMxSliceNum * (*TypeNum) * sizeof(float)) ;
	hipMemcpy( d_T2, T2, SpinMxNum * SpinMxSliceNum * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice ) ;
    
    float *d_Gzgrid = NULL;
    hipMalloc( (void**) &d_Gzgrid, SpinMxNum * SpinMxSliceNum * sizeof(float)) ;
	hipMemcpy( d_Gzgrid, Gzgrid, SpinMxNum * SpinMxSliceNum * sizeof(float), hipMemcpyHostToDevice ) ;
    
    float *d_Gygrid = NULL;
    hipMalloc( (void**) &d_Gygrid, SpinMxNum * SpinMxSliceNum * sizeof(float)) ;
	hipMemcpy( d_Gygrid, Gygrid, SpinMxNum * SpinMxSliceNum * sizeof(float), hipMemcpyHostToDevice ) ;
    
    float *d_Gxgrid = NULL;
    hipMalloc( (void**) &d_Gxgrid, SpinMxNum * SpinMxSliceNum * sizeof(float)) ;
	hipMemcpy( d_Gxgrid, Gxgrid, SpinMxNum * SpinMxSliceNum * sizeof(float), hipMemcpyHostToDevice ) ;
    
    float *d_dB0 = NULL;
    hipMalloc( (void**) &d_dB0, SpinMxNum * SpinMxSliceNum * sizeof(float)) ;
	hipMemcpy( d_dB0, dB0, SpinMxNum * SpinMxSliceNum * sizeof(float), hipMemcpyHostToDevice ) ;

    float *d_TxCoilmg = NULL;
    hipMalloc( (void**) &d_TxCoilmg, SpinMxNum * SpinMxSliceNum * (*TxCoilNum) * sizeof(float)) ;
	hipMemcpy( d_TxCoilmg, TxCoilmg, SpinMxNum * SpinMxSliceNum * (*TxCoilNum) * sizeof(float), hipMemcpyHostToDevice ) ;

    float *d_TxCoilpe = NULL;
    hipMalloc( (void**) &d_TxCoilpe, SpinMxNum * SpinMxSliceNum * (*TxCoilNum) * sizeof(float)) ;
	hipMemcpy( d_TxCoilpe, TxCoilpe, SpinMxNum * SpinMxSliceNum * (*TxCoilNum) * sizeof(float), hipMemcpyHostToDevice ) ;
	
	float *d_RxCoilx = NULL;
    hipMalloc( (void**) &d_RxCoilx, SpinMxNum * SpinMxSliceNum * (*RxCoilNum) * sizeof(float)) ;
	hipMemcpy( d_RxCoilx, RxCoilx, SpinMxNum * SpinMxSliceNum * (*RxCoilNum) * sizeof(float), hipMemcpyHostToDevice ) ;

	float *d_RxCoily = NULL;
    hipMalloc( (void**) &d_RxCoily, SpinMxNum * SpinMxSliceNum * (*RxCoilNum) * sizeof(float)) ;
	hipMemcpy( d_RxCoily, RxCoily, SpinMxNum * SpinMxSliceNum * (*RxCoilNum) * sizeof(float), hipMemcpyHostToDevice ) ;
	
    double *d_CS = NULL;
    hipMalloc( (void**) &d_CS, *TypeNum * sizeof(double)) ;
	hipMemcpy( d_CS, CS, *TypeNum * sizeof(double), hipMemcpyHostToDevice ) ;
	
/* allocate device memory for GPU execution sequence*/
    float *d_Sig = NULL;
    hipMalloc( (void**) &d_Sig, (5+3*(*TxCoilNum)) * MaxutsStep * sizeof(float)) ;
	
/* set CPU signal buffer */
	Sxbuffer    = ippsMalloc_32f(SpinMxNum * PreSignalLen * (*TypeNum) * (*RxCoilNum));
	Sybuffer    = ippsMalloc_32f(SpinMxNum * PreSignalLen * (*TypeNum) * (*RxCoilNum));

/* allocate device memory for buffering acquired signal */
    float *d_Sx = NULL;
    hipMalloc( (void**) &d_Sx, SpinMxNum * PreSignalLen * (*TypeNum) * (*RxCoilNum) * sizeof(float)) ;
    float *d_Sy = NULL;
    hipMalloc( (void**) &d_Sy, SpinMxNum * PreSignalLen * (*TypeNum) * (*RxCoilNum) * sizeof(float)) ;

/* start simulator execution loop */
	mexPrintf("------ Current active GPU device : %s ------\n", &deviceProp.name[0]);
    mexPrintf("TR Counts: %d of %d\n", 1, *TRNum);
    while (i < MaxStep){
        /* check MR sequence pulse flag */
        flag[0]=0;
        flag[1]=0;
        flag[2]=0;
        flag[3]=0;
        flag[4]=0;
        flag[5]=0;
        if (tsLine[i]!=tsLine[i+1]){
            flag[0]+=flagsLine[i*6];
            flag[1]+=flagsLine[i*6+1];
            flag[2]+=flagsLine[i*6+2];
            flag[3]+=flagsLine[i*6+3];
            flag[4]+=flagsLine[i*6+4];
            flag[5]+=flagsLine[i*6+5];
            i++;
        }
        else{
            flag[0]+=flagsLine[i*6];
            flag[1]+=flagsLine[i*6+1];
            flag[2]+=flagsLine[i*6+2];
            flag[3]+=flagsLine[i*6+3];
            flag[4]+=flagsLine[i*6+4];
            flag[5]+=flagsLine[i*6+5];
            
            while (tsLine[i]==tsLine[i+1]){
                flag[0]+=flagsLine[(i+1)*6];
                flag[1]+=flagsLine[(i+1)*6+1];
                flag[2]+=flagsLine[(i+1)*6+2];
                flag[3]+=flagsLine[(i+1)*6+3];
                flag[4]+=flagsLine[(i+1)*6+4];
                flag[5]+=flagsLine[(i+1)*6+5];
                i++;
                if (i==MaxStep-1){
                    break;
                }
            }
            i++;
        }
        
        /* update pulse status */
        *t 	= *(utsLine + *utsi);
        *dt 	= *(utsLine + (int)min(*utsi+1, MaxutsStep-1))-*(utsLine + *utsi);
        *utsi = (int)min(*utsi+1, MaxutsStep-1);
		if (*dt > 0) g_Sig.push_back((float)*dt);
		
        if (flag[0]>=1 ){ /* update rfAmp, rfPhase, rfFreq, rfCoil for multiple rf lines */
            for (j = 0; j < flag[0]; j++){
				 *rfCoil = *(rfCoilLine+ *rfi);
				 TxCoili = (int)(*rfCoil);
				 s = *rfi + 1;
				 while (s < MaxrfStep){
					if (*rfCoil == *(rfCoilLine + s)){
						if (fabs(*(rfAmpLine+ *rfi)) <= fabs(*(rfAmpLine + s)))
							*(rfAmp + TxCoili - 1)= *(rfAmpLine+ *rfi);
						else
							*(rfAmp + TxCoili - 1)= *(rfAmpLine+ s);
	                    
						if (fabs(*(rfPhaseLine+ *rfi)) <= fabs(*(rfPhaseLine + s)))
							*(rfPhase + TxCoili - 1)= *(rfPhaseLine+ *rfi);
						else
							*(rfPhase + TxCoili - 1)= *(rfPhaseLine+ s);
	                    
						if (fabs(*(rfFreqLine+ *rfi)) <= fabs(*(rfFreqLine + s)))
							*(rfFreq + TxCoili - 1)= *(rfFreqLine+ *rfi);
						else
							*(rfFreq + TxCoili - 1)= *(rfFreqLine+ s);
						break;
					}
					s++;
				 }
				 (*rfi)++;
            }
			
			for (j = 0; j < *TxCoilNum; j++){ /* multi-Tx, deal with rfPhase */
				if (rfAmp[j]<0){
					rfAmp[j]=fabs(rfAmp[j]);
					rfPhase[j]=rfPhase[j]+PI;
				}
			}
			
        }
		if (*dt > 0){
			for (j = 0; j < *TxCoilNum; j++){
				g_Sig.push_back((float)rfAmp[j]);
				g_Sig.push_back((float)rfPhase[j]);
				g_Sig.push_back((float)rfFreq[j]);
			}
		}

        if (flag[1]==1 ){ /* update GzAmp */
            if (fabs(*(GzAmpLine+ *Gzi)) <= fabs(*(GzAmpLine + (int)min(*Gzi+1, MaxGzStep-1))))
                *GzAmp = *(GzAmpLine+ *Gzi);
            else
                *GzAmp = *(GzAmpLine+ *Gzi+1);
            
            (*Gzi)++;
        }
        if (*dt > 0) g_Sig.push_back((float)*GzAmp);
		
        if (flag[2]==1 ){ /* update GyAmp */
            if (fabs(*(GyAmpLine+ *Gyi)) <= fabs(*(GyAmpLine + (int)min(*Gyi+1, MaxGyStep-1))))
                *GyAmp = *(GyAmpLine+ *Gyi);
            else
                *GyAmp = *(GyAmpLine+ *Gyi+1);
            
            (*Gyi)++;
        }
        if (*dt > 0) g_Sig.push_back((float)*GyAmp);
		
        if (flag[3]==1 ){ /* update GxAmp */
            if (fabs(*(GxAmpLine+ *Gxi)) <= fabs(*(GxAmpLine + (int)min(*Gxi+1, MaxGxStep-1))))
                *GxAmp = *(GxAmpLine+ *Gxi);
            else
                *GxAmp = *(GxAmpLine+ *Gxi+1);
            
            (*Gxi)++;
        }			
		if (*dt > 0) g_Sig.push_back((float)*GxAmp);
        
        *ADC = 0;   /* avoid ADC overflow */
        if (flag[4]==1){ /* update ADC */
            *ADC = *(ADCLine+ *ADCi);
            (*ADCi)++;
        }
		if (*dt > 0) g_Sig.push_back((float)*ADC);
		
		if (*ADC == 1){
			/* update k-space */
            Kz[Signali] += *KzTmp;
            Ky[Signali] += *KyTmp;
            Kx[Signali] += *KxTmp;
            Signali++;
		}
		
		 /* update Kz, Ky & Kx buffer */
        *KzTmp +=(*GzAmp)*(*dt)*(*Gyro/(2*PI));
        *KyTmp +=(*GyAmp)*(*dt)*(*Gyro/(2*PI));
        *KxTmp +=(*GxAmp)*(*dt)*(*Gyro/(2*PI));
		
        if (flag[5]==1){ /* update Ext */
            *Ext = *(ExtLine+ *Exti);
            /* execute extended process */
            if (*Ext != 0){
				if (g_Sig.size() !=0){
				
					/* calculate signal length */
					SignalLen = Signali-Signalptr;

					/* reset buffer */
					if (PreSignalLen!=SignalLen && SignalLen>0){
						PreSignalLen = SignalLen;
						/* allocate device memory for acquired signal buffer */
						hipFree(d_Sx);
						hipFree(d_Sy);
						hipMalloc( (void**) &d_Sx, SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum) * sizeof(float)) ;
						hipMalloc( (void**) &d_Sy, SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum) * sizeof(float)) ;
						/* zero signal buffer */
						hipMemset(d_Sx, 0 ,SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum) * sizeof(float)); /* only work for 0 */
						hipMemset(d_Sy, 0 ,SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum) * sizeof(float)); /* only work for 0 */
						/* set buffer */
						ippsFree(Sxbuffer);
						ippsFree(Sybuffer);
						Sxbuffer = ippsMalloc_32f(SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum));
						Sybuffer = ippsMalloc_32f(SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum));
					}

					/* avoid shared memory overflow */
					if (g_Sig.size() * sizeof(float) > deviceProp.sharedMemPerBlock){
						SBufferLen = 0;
					}else{
						SBufferLen = g_Sig.size() * sizeof(float);
					}

					/* upload GPU sequence */
					hipMemcpy( d_Sig, 	&g_Sig[0], 	g_Sig.size() * sizeof(float),	hipMemcpyHostToDevice ) ;

					/* call GPU kernel for spin discrete precessing */
					BlochKernelNormalGPU<<< dimGridImg, dimBlockImg, SBufferLen >>>
										((float)*Gyro, d_CS, d_Rho, d_T1, d_T2, d_Mz, d_My, d_Mx,
										d_dB0, d_dWRnd, d_Gzgrid, d_Gygrid, d_Gxgrid, d_TxCoilmg, d_TxCoilpe, d_RxCoilx, d_RxCoily,
										d_Sig, (float)*RxCoilDefault, (float)*TxCoilDefault,
										d_Sx, d_Sy, (float)*rfRef, SignalLen, SBufferLen,
										SpinMxColNum, SpinMxRowNum, SpinMxSliceNum, *SpinNum, *TypeNum, *TxCoilNum, *RxCoilNum, g_Sig.size()/(5+3*(*TxCoilNum)));
					hipDeviceSynchronize();
					g_Sig.clear();
					Signalptr = Signali; /* shift signal array pointer */
				}
				
				/* signal acquisition */
				if (SignalLen>0){
					/* get Sx, Sy buffer from GPU */
					hipMemcpy( Sybuffer, d_Sy, SpinMxNum * SignalLen * (*RxCoilNum) * (*TypeNum) * sizeof(float), hipMemcpyDeviceToHost ) ;
					hipMemcpy( Sxbuffer, d_Sx, SpinMxNum * SignalLen * (*RxCoilNum) * (*TypeNum) * sizeof(float), hipMemcpyDeviceToHost ) ;
					
					/* sum MR signal via openMP */
					for (Typei = 0; Typei < *TypeNum; Typei++){
						for (RxCoili = 0; RxCoili < *RxCoilNum; RxCoili++){  /* signal acquisition per Rx coil */
							#pragma omp parallel
							{   
								#pragma omp for private(j, s, p_Sx, p_Sy, buffer) 
								for (j=0; j < SignalLen; j++){
									
									if (j==0){
										*ActiveThreadNum = omp_get_num_threads();
									}
									
									s=Signali-SignalLen+j;
									p_Sx = Sx + (Typei*(*RxCoilNum)*(*SignalNum)+RxCoili*(*SignalNum)+s);
									p_Sy = Sy + (Typei*(*RxCoilNum)*(*SignalNum)+RxCoili*(*SignalNum)+s);
								
									ippsSum_32f(&Sxbuffer[Typei * (SpinMxNum * SignalLen * (*RxCoilNum)) + RxCoili * (SpinMxNum * SignalLen) +  j*SpinMxNum], SpinMxNum, &buffer, ippAlgHintFast);
									*p_Sx = (double)buffer;
									ippsSum_32f(&Sybuffer[Typei * (SpinMxNum * SignalLen * (*RxCoilNum)) + RxCoili * (SpinMxNum * SignalLen) +  j*SpinMxNum], SpinMxNum, &buffer, ippAlgHintFast);
									*p_Sy = (double)buffer;
								
								}
							}
						}       
					}
					
					/* zero signal buffer */
					hipMemset(d_Sx, 0 ,SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum) * sizeof(float)); /* only work for 0 */
					hipMemset(d_Sy, 0 ,SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum) * sizeof(float)); /* only work for 0 */
				}

			    /* fetch GPU data? */
                ExtCall = mexEvalString("DoGPUFetch");
                if (ExtCall){
                    mexErrMsgTxt("Extended process encounters ERROR!");
                    return;
                }
				
				if (*gpuFetch !=0){
					/* fetch data from GPU */
					hipMemcpy( Mz, d_Mz, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float), hipMemcpyDeviceToHost );
					hipMemcpy( My, d_My, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float), hipMemcpyDeviceToHost );
					hipMemcpy( Mx, d_Mx, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float), hipMemcpyDeviceToHost );
					hipMemcpy( dWRnd, d_dWRnd, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float), hipMemcpyDeviceToHost );
					hipMemcpy( Rho, d_Rho, SpinMxNum * SpinMxSliceNum * (*TypeNum) * sizeof(float), hipMemcpyDeviceToHost );
					hipMemcpy( T1, d_T1, SpinMxNum * SpinMxSliceNum * (*TypeNum) * sizeof(float), hipMemcpyDeviceToHost );
					hipMemcpy( T2, d_T2, SpinMxNum * SpinMxSliceNum * (*TypeNum) * sizeof(float), hipMemcpyDeviceToHost );
					hipMemcpy( Gzgrid, d_Gzgrid, SpinMxNum * SpinMxSliceNum * sizeof(float), hipMemcpyDeviceToHost );
					hipMemcpy( Gygrid, d_Gygrid, SpinMxNum * SpinMxSliceNum * sizeof(float), hipMemcpyDeviceToHost );
					hipMemcpy( Gxgrid, d_Gxgrid, SpinMxNum * SpinMxSliceNum * sizeof(float), hipMemcpyDeviceToHost );
					hipMemcpy( dB0, d_dB0, SpinMxNum * SpinMxSliceNum * sizeof(float), hipMemcpyDeviceToHost );
					hipMemcpy( TxCoilmg, d_TxCoilmg, SpinMxNum * SpinMxSliceNum * (*TxCoilNum) * sizeof(float), hipMemcpyDeviceToHost );
					hipMemcpy( TxCoilpe, d_TxCoilpe, SpinMxNum * SpinMxSliceNum * (*TxCoilNum) * sizeof(float), hipMemcpyDeviceToHost );
					hipMemcpy( RxCoilx, d_RxCoilx, SpinMxNum * SpinMxSliceNum * (*RxCoilNum) * sizeof(float), hipMemcpyDeviceToHost );
					hipMemcpy( RxCoily, d_RxCoily, SpinMxNum * SpinMxSliceNum * (*RxCoilNum) * sizeof(float), hipMemcpyDeviceToHost );
				}

                /* execute extended process */
                ExtCall = mexEvalString("DoExtPlugin");
                if (ExtCall){
                    mexErrMsgTxt("Extended process encounters ERROR!");
                    return;
                }
				
                /* update pointers, avoid pointer change between Matlab and Mex call */
                t               = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "t"));
                dt              = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "dt"));
                rfAmp           = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "rfAmp"));
                rfPhase         = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "rfPhase"));
                rfFreq          = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "rfFreq"));
                rfCoil          = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "rfCoil"));
                rfRef           = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "rfRef"));
                GzAmp           = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "GzAmp"));
                GyAmp           = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "GyAmp"));
                GxAmp           = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "GxAmp"));
                ADC             = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "ADC"));
                Ext             = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "Ext"));
                KzTmp           = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "Kz"));
                KyTmp           = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "Ky"));
                KxTmp           = (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "Kx"));
                gpuFetch     	= (double*) mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "gpuFetch"));
                utsi            = (int*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "utsi"));
                rfi             = (int*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "rfi"));
                Gzi             = (int*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "Gzi"));
                Gyi             = (int*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "Gyi"));
                Gxi             = (int*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "Gxi"));
                ADCi            = (int*)	mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "ADCi"));
                Exti            = (int*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "Exti"));
                TRCount         = (int*)    mxGetData(mxGetField(mexGetVariablePtr("global", "VVar"), 0, "TRCount"));

				if (*gpuFetch !=0){
					*gpuFetch =0;
					/* update pointers, avoid pointer change between Matlab and Mex call */
					Mz          = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VObj"), 0, "Mz"));
					My          = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VObj"), 0, "My"));
					Mx          = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VObj"), 0, "Mx"));
					Rho         = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VObj"), 0, "Rho"));
					T1          = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VObj"), 0, "T1"));
					T2          = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VObj"), 0, "T2"));
					dWRnd       = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VMag"), 0, "dWRnd"));
					dB0         = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VMag"), 0, "dB0"));
					Gzgrid      = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VMag"), 0, "Gzgrid"));
					Gygrid      = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VMag"), 0, "Gygrid"));
					Gxgrid      = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VMag"), 0, "Gxgrid"));
					TxCoilmg    = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VCoi"), 0, "TxCoilmg"));
					TxCoilpe    = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VCoi"), 0, "TxCoilpe"));
					RxCoilx     = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VCoi"), 0, "RxCoilx"));
					RxCoily     = (float*) mxGetData(mxGetField(mexGetVariablePtr("global", "VCoi"), 0, "RxCoily"));

					/* send data back to GPU */
					hipMemcpy( d_Mz, Mz, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice );
					hipMemcpy( d_My, My, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice );
					hipMemcpy( d_Mx, Mx, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice );
					hipMemcpy( d_dWRnd, dWRnd, SpinMxNum * SpinMxSliceNum * (*SpinNum) * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice );
					hipMemcpy( d_Rho, Rho, SpinMxNum * SpinMxSliceNum * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice );
					hipMemcpy( d_T1, T1, SpinMxNum * SpinMxSliceNum * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice );
					hipMemcpy( d_T2, T2, SpinMxNum * SpinMxSliceNum * (*TypeNum) * sizeof(float), hipMemcpyHostToDevice );
					hipMemcpy( d_Gzgrid, Gzgrid, SpinMxNum * SpinMxSliceNum * sizeof(float), hipMemcpyHostToDevice );
					hipMemcpy( d_Gygrid, Gygrid, SpinMxNum * SpinMxSliceNum * sizeof(float), hipMemcpyHostToDevice );
					hipMemcpy( d_Gxgrid, Gxgrid, SpinMxNum * SpinMxSliceNum * sizeof(float), hipMemcpyHostToDevice );
					hipMemcpy( d_dB0, dB0, SpinMxNum * SpinMxSliceNum * sizeof(float), hipMemcpyHostToDevice );
					hipMemcpy( d_TxCoilmg, TxCoilmg, SpinMxNum * SpinMxSliceNum * (*TxCoilNum) * sizeof(float), hipMemcpyHostToDevice );
					hipMemcpy( d_TxCoilpe, TxCoilpe, SpinMxNum * SpinMxSliceNum * (*TxCoilNum) * sizeof(float), hipMemcpyHostToDevice );
					hipMemcpy( d_RxCoilx, RxCoilx, SpinMxNum * SpinMxSliceNum * (*RxCoilNum) * sizeof(float), hipMemcpyHostToDevice );
					hipMemcpy( d_RxCoily, RxCoily, SpinMxNum * SpinMxSliceNum * (*RxCoilNum) * sizeof(float), hipMemcpyHostToDevice );
				}
            }
            (*Exti)++;
        }
        
        if (flag[0]+flag[1]+flag[2]+flag[3]+flag[4]+flag[5] == 0){ /* reset VVar */
            ippsZero_64f(rfAmp, *TxCoilNum);
            ippsZero_64f(rfPhase, *TxCoilNum);
            ippsZero_64f(rfFreq, *TxCoilNum);
            *GzAmp = 0;
            *GyAmp = 0;
            *GxAmp = 0;
            *ADC = 0;
            *Ext = 0;
        }
        
		/* check TR point & end of time point */
		 if (*dt <= 0){ 
			if (g_Sig.size() !=0){
				/* calculate signal length */
				SignalLen = Signali-Signalptr;

				/* reset buffer if needed */
				if (PreSignalLen!=SignalLen && SignalLen>0){
					PreSignalLen = SignalLen;
					/* allocate device memory for acquired signal buffer */
					hipFree(d_Sx);
					hipFree(d_Sy);
					hipMalloc( (void**) &d_Sx, SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum) * sizeof(float)) ;
					hipMalloc( (void**) &d_Sy, SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum) * sizeof(float)) ;
					/* zero signal buffer */
					hipMemset(d_Sx, 0 ,SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum) * sizeof(float)); /* only work for 0 */
					hipMemset(d_Sy, 0 ,SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum) * sizeof(float)); /* only work for 0 */
					/* set buffer */
					ippsFree(Sxbuffer);
					ippsFree(Sybuffer);
					Sxbuffer = ippsMalloc_32f(SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum));
					Sybuffer = ippsMalloc_32f(SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum));
				}

				/* avoid shared memory overflow */
				if (g_Sig.size() * sizeof(float) > deviceProp.sharedMemPerBlock){
					SBufferLen = 0;
				}else{
					SBufferLen = g_Sig.size() * sizeof(float);
				}

				/* upload GPU sequence */
				hipMemcpy( d_Sig, 	&g_Sig[0], 	g_Sig.size() * sizeof(float),	hipMemcpyHostToDevice ) ;

				/* call GPU kernel for spin discrete precessing */
				BlochKernelNormalGPU<<< dimGridImg, dimBlockImg, SBufferLen >>>
									((float)*Gyro, d_CS, d_Rho, d_T1, d_T2, d_Mz, d_My, d_Mx,
									d_dB0, d_dWRnd, d_Gzgrid, d_Gygrid, d_Gxgrid, d_TxCoilmg, d_TxCoilpe, d_RxCoilx, d_RxCoily,
									d_Sig, (float)*RxCoilDefault, (float)*TxCoilDefault,
									d_Sx, d_Sy, (float)*rfRef, SignalLen, SBufferLen,
									SpinMxColNum, SpinMxRowNum, SpinMxSliceNum, *SpinNum, *TypeNum, *TxCoilNum, *RxCoilNum, g_Sig.size()/(5+3*(*TxCoilNum)));
				hipDeviceSynchronize(); /* stablize simulation */
				g_Sig.clear();
				Signalptr = Signali;
			}
			
			/* signal acquisition */
			if (SignalLen>0){
				/* get Sx, Sy buffer from GPU */
				hipMemcpy( Sybuffer, d_Sy, SpinMxNum * SignalLen * (*RxCoilNum) * (*TypeNum) * sizeof(float), hipMemcpyDeviceToHost ) ;
				hipMemcpy( Sxbuffer, d_Sx, SpinMxNum * SignalLen * (*RxCoilNum) * (*TypeNum) * sizeof(float), hipMemcpyDeviceToHost ) ;
				
				/* sum MR signal via openMP */
				for (Typei = 0; Typei < *TypeNum; Typei++){
					for (RxCoili = 0; RxCoili < *RxCoilNum; RxCoili++){  /* signal acquisition per Rx coil */
						#pragma omp parallel
						{   
							#pragma omp for private(j, s, p_Sx, p_Sy, buffer) 
							for (j=0; j < SignalLen; j++){
								
								if (j==0){
									*ActiveThreadNum = omp_get_num_threads();
								}
								
								s=Signali-SignalLen+j;
								p_Sx = Sx + (Typei*(*RxCoilNum)*(*SignalNum)+RxCoili*(*SignalNum)+s);
								p_Sy = Sy + (Typei*(*RxCoilNum)*(*SignalNum)+RxCoili*(*SignalNum)+s);
							
								ippsSum_32f(&Sxbuffer[Typei * (SpinMxNum * SignalLen * (*RxCoilNum)) + RxCoili * (SpinMxNum * SignalLen) +  j*SpinMxNum], SpinMxNum, &buffer, ippAlgHintFast);
								*p_Sx = (double)buffer;
								ippsSum_32f(&Sybuffer[Typei * (SpinMxNum * SignalLen * (*RxCoilNum)) + RxCoili * (SpinMxNum * SignalLen) +  j*SpinMxNum], SpinMxNum, &buffer, ippAlgHintFast);
								*p_Sy = (double)buffer;
							
							}
						}
					}       
				}
				
				/* zero signal buffer */
				hipMemset(d_Sx, 0 ,SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum) * sizeof(float)); /* only work for 0 */
				hipMemset(d_Sy, 0 ,SpinMxNum * SignalLen * (*TypeNum) * (*RxCoilNum) * sizeof(float)); /* only work for 0 */
			}

			if (*dt < 0){
				(*TRCount)++;
				mexPrintf("TR Counts: %d of %d\n", *TRCount, *TRNum);
			}
        }
    }

    /* free GPU memory */
    hipFree(d_Mz);
    hipFree(d_My);
    hipFree(d_Mx);
    hipFree(d_dWRnd);
    hipFree(d_Rho);
    hipFree(d_T1);
    hipFree(d_T2);
    hipFree(d_Gzgrid);
    hipFree(d_Gygrid);
    hipFree(d_Gxgrid);
    hipFree(d_dB0);
    hipFree(d_TxCoilmg);
    hipFree(d_TxCoilpe);
	hipFree(d_RxCoilx);
    hipFree(d_RxCoily);
    hipFree(d_CS);
    hipFree(d_Sig);
	hipFree(d_Sx);
	hipFree(d_Sy);
	
	/* reset device, may slow down subsequent startup due to initialization */
	// hipDeviceReset();
    return 0;
}    

/*DoScanAtGPU.cu(75): warning #951-D: return type of function "main" must be "int"                                          void main(){                                                                                                                 ^
Remark: The warnings can be suppressed with "-diag-suppress <warning-number>"
DoScanAtGPU.cu(76): error: incomplete type is not allowed                                                                     std::ifstream inputFile("/root/output/labeledSpaceJSON/1.pkl.json");                                                                  ^
DoScanAtGPU.cu(84): error: identifier "mwSize" is undefined                                                                   const mwSize *SpinMxDims;                                                                                                     ^
DoScanAtGPU.cu(112): error: "j" has already been declared in the current scope                                                int i=0, j=0, s=0, Signali=0, Signalptr=0, PreSignalLen=0, SignalLen=0, SBufferLen=0, Typei, RxCoili, TxCoili;                   ^
DoScanAtGPU.cu(117): error: identifier "Ipp32f" is undefined                                                                  Ipp32f buffer, *Sxbuffer, *Sybuffer;                                                                                    ^
DoScanAtGPU.cu(128): error: invalid type conversion                                                                          Gyro = (float*) 2.67e08;                                                                                                       ^                                                                                                                                                                                                                                   DoScanAtGPU.cu(135): error: expression must have arithmetic or unscoped enum type                                             SpinNum = (int*) j["xSize"] * j["ySize"];                                                                                         ^                                                                                                                                                                                                                               DoScanAtGPU.cu(139): error: expression must have arithmetic or unscoped enum type                                             dB0 = (float*) j["xSize"] * (int*) j["ySize"] * (int*) j["zSize"];                                                            ^                                                                                                                                                                                                                                   DoScanAtGPU.cu(139): error: expression must have arithmetic or unscoped enum type                                             dB0 = (float*) j["xSize"] * (int*) j["ySize"] * (int*) j["zSize"];                                                                                  ^                                                                                                                                                                                                             DoScanAtGPU.cu(140): error: expression must have arithmetic or unscoped enum type                                             dWRnd = (float*) j["xSize"] * (int*) j["ySize"] * (int*) j["zSize"] * SpinNum * TypeNum;             
                                                                                                                        
                                     ^

DoScanAtGPU.cu(152): error: a value of type "int" cannot be assigned to an entity of type "double *"
   TxCoilDefault = 1;
                 ^

DoScanAtGPU.cu(153): error: a value of type "int" cannot be assigned to an entity of type "double *"
      RxCoilDefault = 1;
                    ^

DoScanAtGPU.cu(222): error: "SpinMxDims" has already been declared in the current scope
      size_t *SpinMxDims = (size_t*) malloc(N * sizeof(size_t));                                                                                                                        
                                                                                                                        
      
      DoScanAtGPU.cu(222): error: identifier "N" is undefined
      size_t *SpinMxDims = (size_t*) malloc(N * sizeof(size_t));
                                            ^

DoScanAtGPU.cu(223): error: identifier "mwSize" is undefined
      SpinMxDims[0] = (mwSize) j["xSize"];
                       ^

DoScanAtGPU.cu(223): error: expected a ";"
      SpinMxDims[0] = (mwSize) j["xSize"];
                               ^

DoScanAtGPU.cu(224): error: expected a ";"
      SpinMxDims[1] = (mwSize) j["ySize"];
                               ^

DoScanAtGPU.cu(225): error: expected a ";"
      SpinMxDims[2] = (mwSize) j["zSize"];
                               ^

DoScanAtGPU.cu(332): error: identifier "ippsMalloc_32f" is undefined
   Sxbuffer = ippsMalloc_32f(SpinMxNum * PreSignalLen * (*TypeNum) * (*RxCoilNum));
              ^

DoScanAtGPU.cu(342): error: identifier "mexPrintf" is undefined
   mexPrintf("------ Current active GPU device : %s ------\n", &deviceProp.name[0]);
   ^

DoScanAtGPU.cu(505): error: identifier "ippsFree" is undefined
        ippsFree(Sxbuffer);
        ^

DoScanAtGPU.cu(555): error: identifier "ippAlgHintFast" is undefined
           ippsSum_32f(&Sxbuffer[Typei * (SpinMxNum * SignalLen * (*RxCoilNum)) + RxCoili * (SpinMxNum * SignalLen) + j*SpinMxNum], SpinMxNum, &buffer, ippAlgHintFast);                                                                                                                                                                                                                                ^                                                                                                                                                                                                               DoScanAtGPU.cu(555): error: identifier "ippsSum_32f" is undefined                                                                  ippsSum_32f(&Sxbuffer[Typei * (SpinMxNum * SignalLen * (*RxCoilNum)) + RxCoili * (SpinMxNum * SignalLen) + j$
      */